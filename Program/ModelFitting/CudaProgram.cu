#include "hip/hip_runtime.h"
							static double M_PI		= 3.141592653589793238462643383279;
	__device__	static double M_PI_d	= 3.141592653589793238462643383279;

	__device__	static unsigned long long	BitList_d[65]={	1,											2,											4,											8,
																												16,											32,											64,											128,
																												256,										512,										1024,										2048,
																												4096,										8192,										16384,									32768,
																												65536,									131072,									262144,									524288,
																												1048576,								2097152,								4194304,								8388608,
																												16777216,								33554432,								67108864,								134217728,
																												268435456,							536870912,							1073741824,							2147483648,
																												4294967296,							8589934592,							17179869184,						34359738368,
																												68719476736,						137438953472,						274877906944,						549755813888,
																												1099511627776,					2199023255552,					4398046511104,					8796093022208,
																												17592186044416,					35184372088832,					70368744177664,					140737488355328,
																												281474976710656,				562949953421312,				1125899906842624,				2251799813685248,
																												4503599627370496,				9007199254740992,				18014398509481984,			36028797018963968,
																												72057594037927936,			144115188075855872,			288230376151711744,			576460752303423488,
																												1152921504606846976,		2305843009213693952,		4611686018427387904,		9223372036854775808	};


							static unsigned long long	BitList[65]={		1,											2,											4,											8,
																												16,											32,											64,											128,
																												256,										512,										1024,										2048,
																												4096,										8192,										16384,									32768,
																												65536,									131072,									262144,									524288,
																												1048576,								2097152,								4194304,								8388608,
																												16777216,								33554432,								67108864,								134217728,
																												268435456,							536870912,							1073741824,							2147483648,
																												4294967296,							8589934592,							17179869184,						34359738368,
																												68719476736,						137438953472,						274877906944,						549755813888,
																												1099511627776,					2199023255552,					4398046511104,					8796093022208,
																												17592186044416,					35184372088832,					70368744177664,					140737488355328,
																												281474976710656,				562949953421312,				1125899906842624,				2251799813685248,
																												4503599627370496,				9007199254740992,				18014398509481984,			36028797018963968,
																												72057594037927936,			144115188075855872,			288230376151711744,			576460752303423488,
																												1152921504606846976,		2305843009213693952,		4611686018427387904,		9223372036854775808	};


							static unsigned char BitSize		= sizeof(unsigned long long	)	*	8;
	__device__	static unsigned char BitSize_d	= sizeof(unsigned long long	)	*	8;


	__device__		unsigned char	List01[7][4]	=	{	{0,2,4,6},{0,1,4,5},{0,1,4,5},{0,1,2,3},{0,1,2,3},{0,1,2,3},{0,1,2,3}	};
	__device__		unsigned char	List02[7][4]	=	{	{1,3,5,7},{2,3,6,7},{3,2,7,6},{4,5,6,7},{5,6,7,4},{6,7,4,5},{7,4,5,6}	};



#include	<stdio.h>
#include	<string.h>
#include	<stdlib.h>
#include	<math.h>
#include	<time.h>
#include	<float.h>
#include	"MT.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>
#include	<windows.h>


	typedef	struct
	{
		unsigned long long	Val[32];
	}	ULL32;


	typedef	struct
	{
		float3	Val[32];
	}	FLO332;


	typedef	struct
	{
		unsigned int				MinMax_Gro;
		unsigned int			 	HideFlag_1;
		unsigned long long	HideData[32];
		float3							LineCo[32];
	}	GROUP_DATA;

	typedef	struct
	{
		unsigned short int	nVertex;
		unsigned short int	nTriangle;
		unsigned short int	nTriangle_32;
		unsigned short int	Kintree_Table[24*2+100];
		ushort3							NewV_Point_32[13792+100];
		ushort3							V_Point3[13776+100];
		ushort3							LandMark01_Vertex[30];
		ushort3							LandMark02_Vertex[30];
		float3							LandMark01_Weight[30];
		float3							LandMark02_Weight[30];
		float								Shape_Dirs[6890*3*300+5000];
		float4							Shape_Dirs4[6890*3*(300/4)+1000];
		int4								Shape_Dirs4_Int[6890*3*(300/4)+1000];
		float								V_Temp[6890*3+100];
		float								J_Regressor[6890*24+100];
		float								Pose_Dirs[6890*3*93+100];
		float3							Pose_Dirs3[6890*3*31+100];
		float								Weights[6890*24+100];
		float4							Weights4[6890*(24/4)+100];

		unsigned short int	J_Reg_Vertex[270];
		unsigned short int	J_Reg_Joint[270];
		float								J_Reg_Co[270];
		unsigned short int	Pose_DirsVer2_Vertex[163534+30];
		unsigned short int	Pose_DirsVer2_FeatITI[163534+30];
		float								Pose_DirsVer2_Co[490602+100];
		unsigned int				Pose_DirsVer2_Start_nFeat[6890*2+100];

		unsigned short int	WeightsVer2_JointITI[18948+100];
		float								WeightsVer2_Co[18948+100];
		unsigned int				WeightsVer2_Start_nFeat[6890*2+100];
	}	STAR_Parameter;


	typedef	struct
	{
		float								TempObje;
		float								TempObje_Betas;
		float								V_Pose[6890*3+100];
		float								V_Pose_Ori[6890*3+100];
		float3							Vertex[6890+10];
		float								Feature[95];
		float								HTrans2[25][4][4];
		float3							GrobalJoint_3[(24)+5];
		float								Model_Joint[24*3+5];

		float2							GrobalJoint_Degi_2_Cam[8][(24)+5];
		float2							Keypoint_Repro_2_Cam[8][(24)+5];
		unsigned long long 	Label_Opt[8][(1080*1920/64)+50];
		GROUP_DATA					GroupData[8][450];
		unsigned char				MinMax_All[8][4];
		unsigned char				ReturnFlag;
	}	STAR_Particle;

	typedef	struct
	{
		unsigned char				nCam;
		unsigned char				nImage;
		unsigned char				nKeyPoint;
		unsigned char				ObjeFlag;
		int									Pix_Yoko;
		int									Pix_Tate;
		int									Pix_Yoko_8;
		int									Pix_Tate_8;
		float								OriUo;
		float								OriVo;
		float								Keypoint[300][45];
		float								KeypointAAA[20][45];
		unsigned long long	Label[300][(1080*1920/64)+50];
		unsigned char				MinMax_All[300][10];
		double							CamPara[10];
		double							CL[3];
		double							Rm[9];
		double							F[1];

	}	Camera_Parameter;

	typedef	struct
	{
		int			nValue;
		int			nPartical;
		int			nPartical_Lo;
		int			nLocal;
		int			Still;
		int			Still02;
	}	OptP;


		double	*mb;
		double	*rb;
		double	*Part_Dis;
		double	*Part_Dis_PB;
		double	*Part_Vel;
		double	*Part_PB;
		double	*Part_Obje;
		int			*Part_Rank;
		double	*Part_NewDis;
		double	*Part_NewVel;


		double	*mb_d;
		double	*rb_d;
		double	*Part_Dis_d;
		double	*Part_Dis_PB_d;
		double	*Part_Vel_d;
		double	*Part_PB_d;
		double	*Part_Obje_d;
		int			*Part_Rank_d;
		double	*Part_NewDis_d;
		double	*Part_NewVel_d;




	OptP				OptPara[1];
	OptP				*OptPara_d;

	hipError_t		ErrorID;

	hiprandStateMtgp32 		*devMTGPStates;
	mtgp32_kernel_params	*devKernelParams;


	STAR_Parameter	*STAR_Para;
	STAR_Parameter	*STAR_Para_d;

	STAR_Particle	*STAR_Partic;
	STAR_Particle	*STAR_Partic_d;


	Camera_Parameter	*Cam_Para;
	Camera_Parameter	*Cam_Para_d;


	void		Get_BasePara(	void	);
	void		Get_BasePara_GPU(	void	);


	#include	"CudaProgram_Obje.h"


	typedef	void	(*Func_Point)(	STAR_Parameter		STAR_Para[],	STAR_Particle		STAR_Partic[],
																Camera_Parameter	Cam_Para[],		OptP						OptPara[],
																float							Part_Dis[],		float						Part_Dis_PB[],
																float							Part_Obje[],	float						Part_PB[],
																float							rb[],					float						mb[]	);

	#include	"CudaProgram_Partical_Swarm.h"

	int	main(void)
	{

/*
    int AAA;    //�f�o�C�X��
   	hipGetDeviceCount(&AAA);

		printf("AA %d \n",	AAA	);

		hipDeviceProp_t dev;
 
		// �f�o�C�X�v���p�e�B�擾
    hipGetDeviceProperties(&dev, 0);

		printf("device %d\n", 0);
		printf(" device name : %s\n", dev.name);
		printf(" total global memory : %d (MB)\n", dev.totalGlobalMem/1024/1024);
		printf(" shared memory / block : %d (KB)\n", dev.sharedMemPerBlock/1024);
		printf(" register / block : %d\n", dev.regsPerBlock);
		printf(" warp size : %d\n", dev.warpSize);
		printf(" max pitch : %d (B)\n", dev.memPitch);
		printf(" max threads / block : %d\n", dev.maxThreadsPerBlock);
		printf(" max size of each dim. of block : (%d, %d, %d)\n", dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
		printf(" max size of each dim. of grid  : (%d, %d, %d)\n", dev.maxGridSize[0], dev.maxGridSize[1], dev.maxGridSize[2]);
		printf(" clock rate : %d (MHz)\n", dev.clockRate/1000);
		printf(" total constant memory : %d (KB)\n", dev.totalConstMem/1024);
		printf(" compute capability : %d.%d\n", dev.major, dev.minor);
		printf(" alignment requirement for texture : %d\n", dev.textureAlignment);
		printf(" device overlap : %s\n", (dev.deviceOverlap ? "ok" : "not"));
		printf(" num. of multiprocessors : %d\n", dev.multiProcessorCount);
		printf(" kernel execution timeout : %s\n", (dev.kernelExecTimeoutEnabled ? "on" : "off"));
		printf(" integrated : %s\n", (dev.integrated ? "on" : "off"));
		printf(" host memory mapping : %s\n", (dev.canMapHostMemory ? "on" : "off"));
		exit(1);

/*
		for (	int iLoop=0;	iLoop<1920;	iLoop++	)
		{
			printf("AA %d %d %d \n",	iLoop,	iLoop%640,	(iLoop%640)/80	);
		}
		exit(1);
*/

		/*--------------------------*/
		/*	Initialize Random Seed	*/
		/*--------------------------*/
		init_genrand((unsigned)time(NULL));

		/*----------------*/
		/*	GPU Memory		*/
		/*----------------*/
		size_t size;
		hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
		printf("Heap Size=%zd\n", size);
		hipDeviceSetLimit(hipLimitMallocHeapSize,	1500000000*1);
		hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
		printf("Heap Size=%zd\n", size);


		/*----------------*/
		/*	About	cuRAND	*/
		/*----------------*/
    hipMalloc(	(void**)&devMTGPStates,	64	*	sizeof(hiprandStateMtgp32)	);

		/* Allocate space MTGP kernel parameters */
		hipMalloc(	(void**)&devKernelParams, sizeof(mtgp32_kernel_params)	);

		/*	Reformat from predefined parameter sets to kernel format		*/
		/*	and copy kernel parameters to device memory									*/
		hiprandMakeMTGP32Constants(	mtgp32dc_params_fast_11213,	devKernelParams	);

		/*	Initialize one state per thread block	*/
    hiprandMakeMTGP32KernelState(devMTGPStates,	mtgp32dc_params_fast_11213,	devKernelParams, 64, genrand_int32()	);

		/*	Generate and use pseudo-random	*/
/*
		generate_kernel<<<1, 10>>>(devMTGPStates);
*/

		/*----------------*/
		/*	Initialize		*/
		/*----------------*/
		FILE		*ID;

		int			i,j,k,n,m,	TempI[10]	=	{0};
		double	TempD[50]	=	{0},	J,	*Q0,	*lb,	*ub;
		float		TempF[50]	=	{0};
		char		TempC[255],	FileName[255];

		WIN32_FIND_DATA		win32fd;
		HANDLE						hFind;
		BOOL							Check_H;

		Func_Point	Func_Hand;




		STAR_Para		=	(STAR_Parameter*)malloc(	sizeof(STAR_Parameter)	*	(1)	);
		hipMalloc(	(void**)&STAR_Para_d,				sizeof(STAR_Parameter)	*	(1)	);


		STAR_Partic		=	(STAR_Particle*)malloc(	sizeof(STAR_Particle)	*	(1024)	);
		hipMalloc(	(void**)&STAR_Partic_d,			sizeof(STAR_Particle)	*	(1024)	);


		Cam_Para		=	(Camera_Parameter*)malloc(	sizeof(Camera_Parameter)	*	(10)	);
		hipMalloc(	(void**)&Cam_Para_d,					sizeof(Camera_Parameter)	*	(10)	);

		/*====================*/
		/*	Get BasePara			*/
		/*====================*/
		Get_BasePara(			);
		printf("%d \n",	1111);

/*
		Get_BasePara_GPU(	);
		printf("%d \n",	1111);
*/

		/*----------------*/
		/*	Check nValue	*/
		/*	�������[�m��	*/
		/*----------------*/
		ID	=	NULL;
		while	(ID==NULL)
		ID	=	fopen(	"./CUDA_Input/Q_Parameter.Dat",	"rb"	);

		i	=	0;
		while	(	1	)
		{
			fseek(ID,	sizeof(double)*i,	SEEK_SET);
			fread(	TempD,	sizeof(double),	1,	ID);

			if	(	TempD[0]	==	pow(10,10)	)
			{
				break;
			}
			i	=	i	+	1;
		}
		fclose(ID);

		/*	�������[�m��	*/
		Q0	=	(double*)malloc(	sizeof(double)	*	(	i	+	40	)	);
		lb	=	(double*)malloc(	sizeof(double)	*	(	i	+	40	)	);
		ub	=	(double*)malloc(	sizeof(double)	*	(	i	+	40	)	);
		TempI[9]	=	i;

		/*----------------------------------*/
		/*	Q_Parameter											*/
		/*	�z��̍Ō�ɔԕ�(DBL_MAX)������	*/
		/*----------------------------------*/
		ID	=	NULL;
		while	(ID==NULL)
		ID	=	fopen(	"./CUDA_Input/Q_Parameter.Dat",	"rb"	);

		i					=	0;
		for (	j=0;	j<3;	j++	)
		{
			TempI[0]	=	0;
			while	(	1	)
			{
				fseek(ID,	sizeof(double)*i,	SEEK_SET);
				fread(	TempD,	sizeof(double),	1,	ID);

				if	(	TempD[0]	==	pow(10,10)	)
				{
					if	(j==0)
					{	Q0[	TempI[0]	]	=	FLT_MAX;	}
					else if (j==1)
					{	lb[	TempI[0]	]	=	FLT_MAX;	}
					else if (j==2)
					{	ub[	TempI[0]	]	=	FLT_MAX;	}
					i	=	i	+	1;
					break;
				}

				if	(j==0)
				{	Q0[	TempI[0]	]	=	TempD[0];	}
				else if (j==1)
				{	lb[	TempI[0]	]	=	TempD[0];	}
				else if (j==2)
				{	ub[	TempI[0]	]	=	TempD[0];	}

				
				TempI[0]				=	TempI[0]	+	1;
				i								=	i	+	1;
			}
		}
		fclose(ID);

/*
		for (	i=0;	i<TempI[9]+1;	i++	)
		{
			if	(	i	<	TempI[9]	)
			{	printf("%d %f %f %f %f\n",i,Q0[i],lb[i],ub[i],	(ub[i]+lb[i])/2	);	}
			else
			{	printf("%d %le %le %le\n",i,Q0[i],lb[i],ub[i]);	}
		}
		exit(1);
*/

		OptPara[0].nPartical		=	1024;



		ParticalSwarm_Initialize(	lb,	ub,	Q0	);
		printf("%d \n",	OptPara[0].nValue);


		ParticalSwarm_Main(	Func_Hand	);











		return(0);
	}



/*======================================================================================================*/
/*	InternalFunction																																										*/
/*======================================================================================================*/
	void	Get_BasePara(	void	)
	{

		double	TempD[30]	=	{0};
		double	S[3],	C[3],	Rm_1[9],	Rm_2[9],	Rm_3[9];
		float		TempF[10]	=	{0.0};

		hipError_t		ErrorID;
		FILE		*ID1;
		int	i,j,k,n,iCam,iSean,iKnown,	iImage,		nCam,	nImage,	nKeyPoint,	ObjeFlag,	LoadStep={0},	TempI[10]	=	{0},	iV,	iU;


		
		unsigned char				*LabelA;
		LabelA	=	(unsigned char*)malloc(sizeof(unsigned char) * ((1080*1920)+100)	);


		int	nValue={0},	nKnownTarm={0},	nUnknownTarm={0},	nValueMat[100][2]={0},	nSean[100]={0},	nSean_Main[100]={0};
		int	iU_8,	iV_8,	iU_Lo,	iV_Lo;




		char	FileName[255],	PathName[255],	LoadName[255];
		char	TempC[5];

		FileName[0]='\0';
		PathName[0]='\0';
		LoadName[0]='\0';

		strcat(	PathName,	"./CUDA_Input/"	);

/*
		printf("%s \n",PathName	);
		exit(1);
*/

		/*==================*/
		/*==================*/
		/*	STAR Parameter	*/
		/*==================*/
		/*==================*/
		STAR_Para[0].nVertex			=	6890;
		STAR_Para[0].nTriangle		=	13776;
		STAR_Para[0].nTriangle_32	=	13792;

		/*================*/
		/*	LandMark			*/
		/*================*/

		//--LandMark01_Vertex
		FileName[0]='\0';
		strcat(	FileName,	"LandMark01_Vertex.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		i					=	0;
		TempI[0]	=	0;
		while	(	1	)
		{
			fseek(	ID1,		sizeof(int)*i,	SEEK_SET			);
			fread(	TempI,	sizeof(int),		1,				ID1	);

			j	=	i	/	3;
			k	=	i	%	3;

//			printf(" %d %d %d %d \n",	i,j,k,TempI[0]-1		);
			if	(	TempI[0]	==	10000	)
			{
				break;
			}

			if			(k==0)
			{
				STAR_Para[0].LandMark01_Vertex[j].x	=	TempI[0]	-	1;
			}
			else if	(k==1)
			{
				STAR_Para[0].LandMark01_Vertex[j].y	=	TempI[0]	-	1;
			}
			else if	(k==2)
			{
				STAR_Para[0].LandMark01_Vertex[j].z	=	TempI[0]	-	1;
			}

			i	=	i	+	1;
		}
		fclose(ID1);



		//--LandMark02_Vertex
		FileName[0]='\0';
		strcat(	FileName,	"LandMark02_Vertex.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		i					=	0;
		TempI[0]	=	0;
		while	(	1	)
		{
			fseek(	ID1,		sizeof(int)*i,	SEEK_SET			);
			fread(	TempI,	sizeof(int),		1,				ID1	);

			j	=	i	/	3;	k	=	i	%	3;

//			printf(" %d %d %d %d \n",	i,j,k,TempI[0]-1		);

			if	(	TempI[0]	==	10000	)
			{
				break;
			}

			if			(k==0)
			{
				STAR_Para[0].LandMark02_Vertex[j].x	=	TempI[0]	-	1;
			}
			else if	(k==1)
			{
				STAR_Para[0].LandMark02_Vertex[j].y	=	TempI[0]	-	1;
			}
			else if	(k==2)
			{
				STAR_Para[0].LandMark02_Vertex[j].z	=	TempI[0]	-	1;
			}

			i	=	i	+	1;
		}
		fclose(ID1);


		//--LandMark01_Weight
		FileName[0]='\0';
		strcat(	FileName,	"LandMark01_Weight.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		i					=	0;
		while	(	1	)
		{
			fseek(	ID1,		sizeof(float)*i,	SEEK_SET			);
			fread(	TempF,	sizeof(float),		1,				ID1	);

			j	=	i	/	3;
			k	=	i	%	3;

//			printf(" %d %d %d %f \n",	i,j,k,TempF[0]		);
			if	(	TempF[0]	==	pow(10,10)	)
			{
				break;
			}

			if			(k==0)
			{
				STAR_Para[0].LandMark01_Weight[j].x	=	TempF[0];
			}
			else if	(k==1)
			{
				STAR_Para[0].LandMark01_Weight[j].y	=	TempF[0];
			}
			else if	(k==2)
			{
				STAR_Para[0].LandMark01_Weight[j].z	=	TempF[0];
			}

			i	=	i	+	1;
		}
		fclose(ID1);



		//--LandMark02_Weight
		FileName[0]='\0';
		strcat(	FileName,	"LandMark02_Weight.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		i					=	0;
		while	(	1	)
		{
			fseek(	ID1,		sizeof(float)*i,	SEEK_SET			);
			fread(	TempF,	sizeof(float),		1,				ID1	);

			j	=	i	/	3;
			k	=	i	%	3;

//			printf(" %d %d %d %f \n",	i,j,k,TempF[0]		);
			if	(	TempF[0]	==	pow(10,10)	)
			{
				break;
			}

			if			(k==0)
			{
				STAR_Para[0].LandMark02_Weight[j].x	=	TempF[0];
			}
			else if	(k==1)
			{
				STAR_Para[0].LandMark02_Weight[j].y	=	TempF[0];
			}
			else if	(k==2)
			{
				STAR_Para[0].LandMark02_Weight[j].z	=	TempF[0];
			}

			i	=	i	+	1;
		}
		fclose(ID1);

//		for (	i=0;	i<17;	i++	)
//		{
//			printf(" %d %d %d %d %f %f %f \n",	i,
//												STAR_Para[0].LandMark01_Vertex[i].x,	STAR_Para[0].LandMark01_Vertex[i].y,	STAR_Para[0].LandMark01_Vertex[i].z,
//												STAR_Para[0].LandMark01_Weight[i].x,	STAR_Para[0].LandMark01_Weight[i].y,	STAR_Para[0].LandMark01_Weight[i].z
//						);
//		}


//		for (	i=0;	i<17;	i++	)
//		{
//			printf(" %d %d %d %d %f %f %f \n",	i,
//												STAR_Para[0].LandMark02_Vertex[i].x,	STAR_Para[0].LandMark02_Vertex[i].y,	STAR_Para[0].LandMark02_Vertex[i].z,
//												STAR_Para[0].LandMark02_Weight[i].x,	STAR_Para[0].LandMark02_Weight[i].y,	STAR_Para[0].LandMark02_Weight[i].z
//						);
//		}
//		exit(1);

		/*==================*/
		/*	Shape_Dire			*/
		/*==================*/
//		TempI[0]	=	(	STAR_Para[0].nVertex*3*300	);
//
//		FileName[0]='\0';
//		strcat(	FileName,	"Shape_Dirs.Dat"	);
//
//		LoadName[0]='\0';
//		strcat(	LoadName,	PathName	);
//		strcat(	LoadName,	FileName	);
//
//		ID1	=	NULL;
//		while	(ID1==NULL)
//		ID1	=	fopen(	LoadName,	"rb"	);
//
//		/*------------------*/
//		/*	�f�[�^�ǂݎ��	*/
//		/*------------------*/
//		fread(	STAR_Para[0].Shape_Dirs,	sizeof(float),	TempI[0],	ID1);
//		fclose(ID1);
//
//		for (	i=0;	i<STAR_Para[0].nVertex*3*75;	i++	)
//		{
//			STAR_Para[0].Shape_Dirs4[i].x	=	STAR_Para[0].Shape_Dirs[(i*4)+0];
//			STAR_Para[0].Shape_Dirs4[i].y	=	STAR_Para[0].Shape_Dirs[(i*4)+1];
//			STAR_Para[0].Shape_Dirs4[i].z	=	STAR_Para[0].Shape_Dirs[(i*4)+2];
//			STAR_Para[0].Shape_Dirs4[i].w	=	STAR_Para[0].Shape_Dirs[(i*4)+3];
//		}
//

//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].Shape_Dirs[i],	STAR_Para[0].Shape_Dirs[TempI[0]-1-i]	);
//		}
//		exit(1);


		/*==================*/
		/*	Shape_Dire			*/
		/*==================*/
		TempI[0]	=	6201600;

		FileName[0]='\0';
		strcat(	FileName,	"Shape_DirsVer3.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].Shape_Dirs,	sizeof(float),	TempI[0],	ID1	);
		fclose(ID1);

		for (	i=0;	i<(TempI[0]/4);	i++	)
		{
			STAR_Para[0].Shape_Dirs4[i].x	=	STAR_Para[0].Shape_Dirs[(i*4)+0];
			STAR_Para[0].Shape_Dirs4[i].y	=	STAR_Para[0].Shape_Dirs[(i*4)+1];
			STAR_Para[0].Shape_Dirs4[i].z	=	STAR_Para[0].Shape_Dirs[(i*4)+2];
			STAR_Para[0].Shape_Dirs4[i].w	=	STAR_Para[0].Shape_Dirs[(i*4)+3];
		}

//		for (	i=0;	i<25;	i++	)
//		{
//			printf(" %d %.15f %.15f \n",	i*4+0,	STAR_Para[0].Shape_Dirs4[i].x,	STAR_Para[0].Shape_Dirs4[(TempI[0]/4)-1-i].w	);
//			printf(" %d %.15f %.15f \n",	i*4+1,	STAR_Para[0].Shape_Dirs4[i].y,	STAR_Para[0].Shape_Dirs4[(TempI[0]/4)-1-i].z	);
//			printf(" %d %.15f %.15f \n",	i*4+2,	STAR_Para[0].Shape_Dirs4[i].z,	STAR_Para[0].Shape_Dirs4[(TempI[0]/4)-1-i].y	);
//			printf(" %d %.15f %.15f \n",	i*4+3,	STAR_Para[0].Shape_Dirs4[i].w,	STAR_Para[0].Shape_Dirs4[(TempI[0]/4)-1-i].x	);
//		}
//		exit(1);



		/*==================*/
		/*	Shape_Dire			*/
		/*==================*/
		TempI[0]	=	6201600;

		FileName[0]='\0';
		strcat(	FileName,	"Shape_DirsVer2.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].Shape_Dirs,	sizeof(float),	TempI[0],	ID1	);
		fclose(ID1);

//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].Shape_Dirs[i],	STAR_Para[0].Shape_Dirs[TempI[0]-1-i]	);
//		}
//		exit(1);


		/*==================*/
		/*	V_Temp					*/
		/*==================*/
		TempI[0]	=	(	STAR_Para[0].nVertex*3	);

		FileName[0]='\0';
		strcat(	FileName,	"V_Temp.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].V_Temp,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);

/*
		for (	i=0;	i<100;	i++	)
		{
			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].V_Temp[i],	STAR_Para[0].V_Temp[TempI[0]-1-i]	);
		}
		exit(1);
*/

		/*==================*/
		/*	J_Regressor			*/
		/*==================*/
		TempI[0]	=	(	STAR_Para[0].nVertex*24	);

		FileName[0]='\0';
		strcat(	FileName,	"J_Regressor.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].J_Regressor,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);




		/*====================*/
		/*	J_Regressor_Ver2	*/
		/*====================*/

		/*--Coeff--*/
		TempI[0]	=	(	260	);

		FileName[0]='\0';
		strcat(	FileName,	"J_RegressorVer2_Co.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].J_Reg_Co,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);


		/*---------*/
		/*--Joint--*/
		/*---------*/
		TempI[0]	=	(	260	);

		FileName[0]='\0';
		strcat(	FileName,	"J_RegressorVer2_Joint.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].J_Reg_Joint,	sizeof(int),	TempI[0],	ID1);
		fclose(ID1);



		/*---------*/
		/*--Joint--*/
		/*---------*/
		TempI[0]	=	(	260	);

		FileName[0]='\0';
		strcat(	FileName,	"J_RegressorVer2_Vertex.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].J_Reg_Vertex,	sizeof(int),	TempI[0],	ID1);
		fclose(ID1);



//		for (	i=0;	i<260;	i++	)
//		{
//			printf(" %d %d %d %.15f \n",	i,	STAR_Para[0].J_Reg_Joint[i],	STAR_Para[0].J_Reg_Vertex[i],	STAR_Para[0].J_Reg_Co[i]	);
//		}
//		exit(1);

/*
		for (	i=0;	i<100;	i++	)
		{
			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].J_Regressor[i],	STAR_Para[0].J_Regressor[TempI[0]-1-i]	);
		}
		exit(1);
*/

		/*==================*/
		/*	Pose_Dirs				*/
		/*==================*/
		TempI[0]	=	(	STAR_Para[0].nVertex*3*93	);

		FileName[0]='\0';
		strcat(	FileName,	"Pose_Dirs.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].Pose_Dirs,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);

		for (	i=0;	i<STAR_Para[0].nVertex*3*(93/3);	i++	)
		{
			STAR_Para[0].Pose_Dirs3[i].x	=	STAR_Para[0].Pose_Dirs[(i*3)+0];
			STAR_Para[0].Pose_Dirs3[i].y	=	STAR_Para[0].Pose_Dirs[(i*3)+1];
			STAR_Para[0].Pose_Dirs3[i].z	=	STAR_Para[0].Pose_Dirs[(i*3)+2];
		}

/*
		for (	i=0;	i<100;	i++	)
		{
			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].Pose_Dirs[i],	STAR_Para[0].Pose_Dirs[TempI[0]-1-i]	);
		}
		exit(1);
*/



		/*==================*/
		/*	Pose Dire_Ver2	*/
		/*==================*/
		/*--Coeff--*/
		TempI[0]	=	(	490602+1	);

		FileName[0]='\0';
		strcat(	FileName,	"Pose_DirsVer2_Co.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].Pose_DirsVer2_Co,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);

//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].Pose_DirsVer2_Co[i],	STAR_Para[0].Pose_DirsVer2_Co[TempI[0]-1-i]	);
//		}
//		exit(1);



		/*---------*/
		/*--Joint--*/
		/*---------*/
		TempI[0]	=	(	163534	+	1	);

		FileName[0]='\0';
		strcat(	FileName,	"Pose_DirsVer2_FeatITI.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].Pose_DirsVer2_FeatITI,	sizeof(unsigned short),	TempI[0],	ID1);
		fclose(ID1);


		/*---------*/
		/*--Joint--*/
		/*---------*/
		TempI[0]	=	(	163534	+	1	);

		FileName[0]='\0';
		strcat(	FileName,	"Pose_DirsVer2_Vertex.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].Pose_DirsVer2_Vertex,	sizeof(unsigned short),	TempI[0],	ID1);
		fclose(ID1);


//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %d %d %d %d \n",	i,	STAR_Para[0].Pose_DirsVer2_Vertex[i],							STAR_Para[0].Pose_DirsVer2_FeatITI[i],
//																				STAR_Para[0].Pose_DirsVer2_Vertex[TempI[0]-1-i],	STAR_Para[0].Pose_DirsVer2_FeatITI[TempI[0]-1-i]	);
//		}
//		exit(1);
//

		/*---------------*/
		/*--Start_nFeat--*/
		/*---------------*/
		TempI[0]	=	(	(6890*2)+1	);

		FileName[0]='\0';
		strcat(	FileName,	"Pose_DirsVer2_Start_nFeat.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].Pose_DirsVer2_Start_nFeat,	sizeof(int),	TempI[0],	ID1);
		fclose(ID1);


//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %d %d %d %d \n",	i,	STAR_Para[0].Pose_DirsVer2_Start_nFeat[i*2+0],				STAR_Para[0].Pose_DirsVer2_Start_nFeat[i*2+1],
//																				STAR_Para[0].Pose_DirsVer2_Start_nFeat[(6890-i)*2-2],	STAR_Para[0].Pose_DirsVer2_Start_nFeat[(6890-i)*2-1]	);
//		}
//		printf(" END %d\n",	STAR_Para[0].Pose_DirsVer2_Start_nFeat[6890*2]	);
//		exit(1);
//

		/*==================*/
		/*	Pose Dire_Ver2	*/
		/*==================*/
		/*--Coeff--*/
		TempI[0]	=	(	18948+1	);

		FileName[0]='\0';
		strcat(	FileName,	"WeightVer2_Co.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].WeightsVer2_Co,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);

//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].WeightsVer2_Co[i],	STAR_Para[0].WeightsVer2_Co[TempI[0]-1-i]	);
//		}
//		exit(1);

		/*---------*/
		/*--Joint--*/
		/*---------*/
		TempI[0]	=	(	18948+1	);

		FileName[0]='\0';
		strcat(	FileName,	"WeightVer2_JointITI.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].WeightsVer2_JointITI,	sizeof(unsigned short),	TempI[0],	ID1);
		fclose(ID1);

//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %d %d \n",	i,	STAR_Para[0].WeightsVer2_JointITI[i],	STAR_Para[0].WeightsVer2_JointITI[TempI[0]-1-i]	);
//		}
//		exit(1);


		/*---------------*/
		/*--Start_nFeat--*/
		/*---------------*/
		TempI[0]	=	(	(6890*2)+1	);

		FileName[0]='\0';
		strcat(	FileName,	"WeightVer2_Start_nFeat.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*	�f�[�^�ǂݎ��	*/
		fread(	STAR_Para[0].WeightsVer2_Start_nFeat,	sizeof(int),	TempI[0],	ID1);
		fclose(ID1);


//		for (	i=0;	i<100;	i++	)
//		{
//			printf(" %d %d %d %d %d \n",	i,	STAR_Para[0].WeightsVer2_Start_nFeat[i*2+0],				STAR_Para[0].WeightsVer2_Start_nFeat[i*2+1],
//																				STAR_Para[0].WeightsVer2_Start_nFeat[(6890-i)*2-2],	STAR_Para[0].WeightsVer2_Start_nFeat[(6890-i)*2-1]	);
//		}
//		printf(" END %d\n",	STAR_Para[0].WeightsVer2_Start_nFeat[6890*2]	);
//		exit(1);

		/*==================*/
		/*	Weights					*/
		/*==================*/
		TempI[0]	=	(	STAR_Para[0].nVertex*24	);

		FileName[0]='\0';
		strcat(	FileName,	"Weights.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].Weights,	sizeof(float),	TempI[0],	ID1);
		fclose(ID1);

		for (	i=0;	i<STAR_Para[0].nVertex*6;	i++	)
		{
			STAR_Para[0].Weights4[i].x	=	STAR_Para[0].Weights[(i*4)+0];
			STAR_Para[0].Weights4[i].y	=	STAR_Para[0].Weights[(i*4)+1];
			STAR_Para[0].Weights4[i].z	=	STAR_Para[0].Weights[(i*4)+2];
			STAR_Para[0].Weights4[i].w	=	STAR_Para[0].Weights[(i*4)+3];
		}


/*
		for (	i=0;	i<100;	i++	)
		{
			printf(" %d %.15f %.15f \n",	i,	STAR_Para[0].Weights[i],	STAR_Para[0].Weights[TempI[0]-1-i]	);
		}
		exit(1);
*/

		/*==================*/
		/*	V_Point					*/
		/*==================*/
		FileName[0]='\0';
		strcat(	FileName,	"V_Point.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		i					=	0;
		TempI[0]	=	0;
		while	(	1	)
		{
			fseek(	ID1,		sizeof(unsigned short int)*i,	SEEK_SET			);
			fread(	TempI,	sizeof(unsigned short int),		1,				ID1	);

			j	=	i	/	3;
			k	=	i	%	3;

			if	(	TempI[0]	==	20000	)
			{
				break;
			}

			if			(k==0)
			{
				STAR_Para[0].V_Point3[j].x	=	TempI[0];
			}
			else if	(k==1)
			{
				STAR_Para[0].V_Point3[j].y	=	TempI[0];
			}
			else if	(k==2)
			{
				STAR_Para[0].V_Point3[j].z	=	TempI[0];
			}

			i	=	i	+	1;
		}
		fclose(ID1);

//		printf(" %d %d \n",	i,j	);
//		exit(1);

/*
		for (	i=0;	i<100;	i++	)
		{
			printf(" %d %d %d %d %d %d \n",	STAR_Para[0].V_Point[i*3+0],	STAR_Para[0].V_Point[i*3+1],	STAR_Para[0].V_Point[i*3+2],
																			STAR_Para[0].V_Point3[i].x,		STAR_Para[0].V_Point3[i].y,		STAR_Para[0].V_Point3[i].z	);
		}
		exit(1);
*/

	/*==================*/
		/*	NewV_Point32		*/
		/*==================*/
		FileName[0]='\0';
		strcat(	FileName,	"V_Point_32.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		i					=	0;
		TempI[0]	=	0;
		while	(	1	)
		{
			fseek(	ID1,		sizeof(unsigned short int)*i,	SEEK_SET			);
			fread(	TempI,	sizeof(unsigned short int),		1,				ID1	);

			j	=	i	/	3;
			k	=	i	%	3;

			if	(	TempI[0]	==	20000	)
			{
				break;
			}

			if			(k==0)
			{
				STAR_Para[0].NewV_Point_32[j].x	=	TempI[0];
			}
			else if	(k==1)
			{
				STAR_Para[0].NewV_Point_32[j].y	=	TempI[0];
			}
			else if	(k==2)
			{
				STAR_Para[0].NewV_Point_32[j].z	=	TempI[0];
			}

			i	=	i	+	1;
		}
		fclose(ID1);

//		for (	i=0;	i<32;	i++	)
//		{
//			printf(" %d %d %d %d \n",	i,	STAR_Para[0].NewV_Point_32[i].x,	STAR_Para[0].NewV_Point_32[i].y,	STAR_Para[0].NewV_Point_32[i].z	);
//		}

//		for (	i=STAR_Para[0].nTriangle_32 - 32;	i<STAR_Para[0].nTriangle_32;	i++	)
//		{
//			printf(" %d %d %d %d \n",	i,	STAR_Para[0].NewV_Point_32[i].x,	STAR_Para[0].NewV_Point_32[i].y,	STAR_Para[0].NewV_Point_32[i].z	);
//		}
//		exit(1);

		/*==================*/
		/*	Kintree_Table		*/
		/*==================*/
		TempI[0]	=	(	24*2	);

		FileName[0]='\0';
		strcat(	FileName,	"Kintree_Table.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	STAR_Para[0].Kintree_Table,	sizeof(unsigned short int),	TempI[0],	ID1);
		fclose(ID1);

/*
		for (	i=0;	i<48;	i++	)
		{
			printf(" %d %d \n",	i,	STAR_Para[0].Kintree_Table[i]	);
		}
		exit(1);
*/


		/*==============*/
		/*	BasePara		*/
		/*==============*/
		FileName[0]='\0';
		strcat(	FileName,	"BasePara_Int.Dat"	);

		LoadName[0]='\0';
		strcat(	LoadName,	PathName	);
		strcat(	LoadName,	FileName	);

		ID1	=	NULL;
		while	(ID1==NULL)
		ID1	=	fopen(	LoadName,	"rb"	);

		/*------------------*/
		/*	�f�[�^�ǂݎ��	*/
		/*------------------*/
		fread(	&nCam,			sizeof(int),	1,	ID1);
		fread(	&nImage,		sizeof(int),	1,	ID1);
		fread(	&nKeyPoint,	sizeof(int),	1,	ID1);
		fread(	&ObjeFlag,	sizeof(int),	1,	ID1);

		for (	iCam=0;	iCam<nCam;	iCam++	)
		{
			Cam_Para[iCam].nCam				=	nCam;
			Cam_Para[iCam].nImage			=	nImage;
			Cam_Para[iCam].nKeyPoint	=	nKeyPoint;
			Cam_Para[iCam].ObjeFlag		=	ObjeFlag;

			fread(	TempI,		sizeof(int),	2,	ID1);

			Cam_Para[iCam].Pix_Tate	=	TempI[0];
			Cam_Para[iCam].Pix_Yoko	=	TempI[1];

//			printf("%d %d %d %d %d %d \n",	Cam_Para[iCam].nCam,	Cam_Para[iCam].nImage,	Cam_Para[iCam].nKeyPoint,	Cam_Para[iCam].ObjeFlag,	Cam_Para[iCam].Pix_Tate,	Cam_Para[iCam].Pix_Yoko	);
		}
		fclose(ID1);


		/*====================*/
		/*	Camera Parameter	*/
		/*====================*/
		for (	iCam=0;	iCam<Cam_Para[0].nCam;	iCam++	)
		{

			Cam_Para[iCam].Pix_Yoko_8	=	Cam_Para[iCam].Pix_Yoko/8;
			Cam_Para[iCam].Pix_Tate_8	=	Cam_Para[iCam].Pix_Tate/8;

			Cam_Para[iCam].OriUo		=	(Cam_Para[iCam].Pix_Yoko+1)/2.00;
			Cam_Para[iCam].OriVo		=	(Cam_Para[iCam].Pix_Tate+1)/2.00;

			/*	CameraParameter	*/
			FileName[0]='\0';
			strcat(	FileName,	"CamPara_Cam0.Dat"	);

			sprintf(TempC,	"%d",	iCam+1	);

			FileName[11]	=	TempC[0];
/*
			printf(" %s \n",	FileName	);
*/
			LoadName[0]='\0';
			strcat(	LoadName,	PathName	);
			strcat(	LoadName,	FileName	);

			ID1	=	NULL;
			while	(ID1==NULL)
			ID1	=	fopen(	LoadName,	"rb"	);

			/*------------------*/
			/*	�f�[�^�ǂݎ��	*/
			/*------------------*/
			fread(	Cam_Para[iCam].CamPara,	sizeof(double),	7,	ID1);
			fclose(ID1);

			/*	Focal	0		*/
			Cam_Para[iCam].F[0]	=	Cam_Para[iCam].CamPara[0];

			/*	CL		1:3	*/
			Cam_Para[iCam].CL[0]	=	Cam_Para[iCam].CamPara[1];
			Cam_Para[iCam].CL[1]	=	Cam_Para[iCam].CamPara[2];
			Cam_Para[iCam].CL[2]	=	Cam_Para[iCam].CamPara[3];

			/*	Angle	4:6	*/
			S[0]	=	sin(	Cam_Para[iCam].CamPara[4]	);
			S[1]	=	sin(	Cam_Para[iCam].CamPara[5]	);
			S[2]	=	sin(	Cam_Para[iCam].CamPara[6]	);

			C[0]	=	cos(	Cam_Para[iCam].CamPara[4]	);
			C[1]	=	cos(	Cam_Para[iCam].CamPara[5]	);
			C[2]	=	cos(	Cam_Para[iCam].CamPara[6]	);

			/*	JRm_1	*/
			Rm_1[0]	=	C[1];		Rm_1[1]	=	S[1];		Rm_1[2]	=	0;
			Rm_1[3]	=	-S[1];	Rm_1[4]	=	C[1];		Rm_1[5]	=	0;
			Rm_1[6]	=	0;			Rm_1[7]	=	0;			Rm_1[8]	=	1;

			/*	JRm_2	*/
			Rm_2[0]	=	1;			Rm_2[1]	=	0;			Rm_2[2]	=	0;
			Rm_2[3]	=	0;			Rm_2[4]	=	C[0];		Rm_2[5]	=	S[0];
			Rm_2[6]	=	0;			Rm_2[7]	=	-S[0];	Rm_2[8]	=	C[0];

			for (	i=0;	i	<	3;	i++	)
			{
				for (	j=0;	j	<	3;	j++	)
				{
					k	=	(i*3)	+	j;
					Rm_3[k]	=
						(	Rm_1[j]		*	Rm_2[(i*3)+0]	)
					+	(	Rm_1[j+3]	*	Rm_2[(i*3)+1]	)
					+	(	Rm_1[j+6]	*	Rm_2[(i*3)+2]	);
				}
			}

			/*	JRm_3	*/
			Rm_1[0]	=	C[2];		Rm_1[1]	=	0;	Rm_1[2]	=	-S[2];
			Rm_1[3]	=	0;			Rm_1[4]	=	1;	Rm_1[5]	=	0;
			Rm_1[6]	=	S[2];		Rm_1[7]	=	0;	Rm_1[8]	=	C[2];

			for (	i=0;	i	<	3;	i++	)
			{
				for (	j=0;	j	<	3;	j++	)
				{
					k	=	(i*3)	+	j;
					Rm_2[k]	=
						(	Rm_3[j]		*	Rm_1[(i*3)+0]	)
					+	(	Rm_3[j+3]	*	Rm_1[(i*3)+1]	)
					+	(	Rm_3[j+6]	*	Rm_1[(i*3)+2]	);
				}
			}

			Cam_Para[iCam].Rm[0]	=	-Rm_2[0];			Cam_Para[iCam].Rm[1]	=	-Rm_2[1];			Cam_Para[iCam].Rm[2]	=	-Rm_2[2];
			Cam_Para[iCam].Rm[3]	=	Rm_2[6];			Cam_Para[iCam].Rm[4]	=	Rm_2[7];			Cam_Para[iCam].Rm[5]	=	Rm_2[8];
			Cam_Para[iCam].Rm[6]	=	Rm_2[3];			Cam_Para[iCam].Rm[7]	=	Rm_2[4];			Cam_Para[iCam].Rm[8]	=	Rm_2[5];
		}

		/*======================*/
		/*	Label	and KeyPoint	*/
		/*======================*/
		for (	iCam=0;	iCam<Cam_Para[0].nCam;	iCam++	)
		{
			for (	iImage=0;	iImage<Cam_Para[0].nImage;	iImage++	)
			{

				/*	Label	*/
				FileName[0]='\0';
				strcat(	FileName,	"Label_Cam0_Image000.Dat"	);

				/*	iCam	*/
				sprintf(TempC,	"%d",	iCam+1	);
				FileName[9]	=	TempC[0];

				/*	iImage	*/
				sprintf(TempC,	"%03d",	iImage+1	);
				FileName[16]	=	TempC[0];
				FileName[17]	=	TempC[1];
				FileName[18]	=	TempC[2];

				LoadName[0]='\0';
				strcat(	LoadName,	PathName	);
				strcat(	LoadName,	FileName	);

				ID1	=	NULL;
				while	(ID1==NULL)
				ID1	=	fopen(	LoadName,	"rb"	);

				/*------------------*/
				/*	�f�[�^�ǂݎ��	*/
				/*------------------*/
				fread(	LabelA,	sizeof(unsigned char),	1080*1920,	ID1);
				fclose(ID1);

				//	Initialize
				for (	i=0;	i<(1080*1920/64);	i++	)
				{
					Cam_Para[iCam].Label[iImage][i]	=	0;
				}

				for (	iU=0;	iU<1920;	iU++	)
				{

					for (	iV=0;	iV<1080;	iV++	)
					{

						iU_8	=	iU/8;	iV_8	=	iV/8;
						iU_Lo	=	iU%8;	iV_Lo	=	iV%8;


						j	=	(iV_8)	+	(	iU_8*135	);
						k	=	(iV_Lo)	+	(	iU_Lo*8		);

						i	=	(iV)	+	(	(iU)*1080	);

						TempI[0]	=	(	i	/	BitSize	);
						TempI[1]	=	(	i	%	BitSize	);
						if	(	LabelA[i]	==	1	)
						{
							Cam_Para[iCam].Label[iImage][	j	]	=	Cam_Para[iCam].Label[iImage][	j	]	|	BitList[	k	];
						}
					}
				}

				//	Cam_Para[iCam].MinMax_All[iImage][0]	=	MaxU_All;
				//	Cam_Para[iCam].MinMax_All[iImage][1]	=	MinU_All;
				//	Cam_Para[iCam].MinMax_All[iImage][2]	=	MaxV_All;
				//	Cam_Para[iCam].MinMax_All[iImage][3]	=	MinV_All;
				//--MaxU_All
				for (	i=0;	i<(240*135);	i++	)
				{
					iU_8	=	239-(i/135);		iV_8	=	i%135;
					j			=	(iU_8*135)	+	iV_8;
					if	(	Cam_Para[iCam].Label[iImage][j]	!=	0)
					{
						Cam_Para[iCam].MinMax_All[iImage][0]	=	iU_8;
						break;
					}
				}

				//--MinU_All
				for (	i=0;	i<(240*135);	i++	)
				{
					iU_8	=	i/135;		iV_8	=	i%135;
					j			=	(iU_8*135)	+	iV_8;
					if	(	Cam_Para[iCam].Label[iImage][j]	!=	0)
					{
						Cam_Para[iCam].MinMax_All[iImage][1]	=	iU_8;
						break;
					}
				}

				//--MaxV_All
				for (	i=0;	i<(240*135);	i++	)
				{
					iU_8	=	i%240;		iV_8	=	134-(i/240);
					j			=	(iU_8*135)	+	iV_8;
					if	(	Cam_Para[iCam].Label[iImage][j]	!=	0)
					{
						Cam_Para[iCam].MinMax_All[iImage][2]	=	iV_8;
						break;
					}
				}

				//--MinV_All
				for (	i=0;	i<(240*135);	i++	)
				{
					iU_8	=	i%240;		iV_8	=	i/240;
					j			=	(iU_8*135)	+	iV_8;
					if	(	Cam_Para[iCam].Label[iImage][j]	!=	0)
					{
						Cam_Para[iCam].MinMax_All[iImage][3]	=	iV_8;
						break;
					}
				}

				/*------------------*/
				/*	�f�[�^�ǂݎ��	*/
				/*------------------*/
				FileName[0]='\0';
				strcat(	FileName,	"Keypoint_Cam0_Image000.Dat"	);

				/*	iCam	*/
				sprintf(TempC,	"%d",	iCam+1	);
				FileName[12]	=	TempC[0];

				/*	iImage	*/
				sprintf(TempC,	"%03d",	iImage+1	);
				FileName[19]	=	TempC[0];
				FileName[20]	=	TempC[1];
				FileName[21]	=	TempC[2];

				LoadName[0]='\0';
				strcat(	LoadName,	PathName	);
				strcat(	LoadName,	FileName	);

				ID1	=	NULL;
				while	(ID1==NULL)
				ID1	=	fopen(	LoadName,	"rb"	);

				/*------------------*/
				/*	�f�[�^�ǂݎ��	*/
				/*------------------*/
				fread(	Cam_Para[iCam].Keypoint[iImage],	sizeof(float),	35,	ID1);
				fclose(ID1);

//				for (	i=0;	i<17;	i++	)
//				{
//					printf("%d %d %d %f %f \n",	iCam,	iImage,	i,	Cam_Para[iCam].Keypoint[iImage][i*2+0],	Cam_Para[iCam].Keypoint[iImage][i*2+1]	);
//				}
//				printf("End %f \n",	Cam_Para[iCam].Keypoint[iImage][34]	);
			}
		}
//		exit(1);

//		for (	iCam=0;	iCam<8;	iCam++	)
//		{
//			for (	iImage=0;	iImage<Cam_Para[0].nImage;	iImage++	)
//			{
//
//				printf("%d %d %d %d %d %d \n",	iCam,	iImage,	Cam_Para[iCam].MinMax_All[iImage][0],	
//																											Cam_Para[iCam].MinMax_All[iImage][1],
//																											Cam_Para[iCam].MinMax_All[iImage][2],
//																											Cam_Para[iCam].MinMax_All[iImage][3]	);
//			}
//		}
//		exit(1);
//

//		for (	iCam=0;	iCam<nCam;	iCam++	)
//		{
//			for (	iImage=0;	iImage<Cam_Para[0].nImage;	iImage++	)
//			{
//				for (	i=0;	i<17;	i++	)
//				{
//					printf("%d %d %d %f %f \n",	iCam,	iImage,	i,	Cam_Para[iCam].Keypoint[iImage][i*2+0],	Cam_Para[iCam].Keypoint[iImage][i*2+1]	);
//				}
//				printf("End %f \n",	Cam_Para[iCam].Keypoint[iImage][34]	);
//			}
//		}
//		exit(1);




		/*------------------*/
		/*	GPUMemory copy	*/
		/*------------------*/
		/*	STAR Parameter	*/
		ErrorID	=	hipMemcpy(	STAR_Para_d,	STAR_Para,	sizeof(STAR_Parameter)	*	1,	hipMemcpyHostToDevice);
		if	(ErrorID	!=	hipSuccess)
		{
			printf(	"\n   CudaMemoryCopy Error!!! 001"	);
			exit(1);
		}

		/*	Camera Parameter	*/
		ErrorID	=	hipMemcpy(	Cam_Para_d,	Cam_Para,	sizeof(Camera_Parameter)	*	Cam_Para[0].nCam,	hipMemcpyHostToDevice);
		if	(ErrorID	!=	hipSuccess)
		{
			printf(	"\n   CudaMemoryCopy Error!!! 001"	);
			exit(1);
		}

		return;	
	}



